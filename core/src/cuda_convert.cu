#include "hip/hip_runtime.h"
// Copyright 2022 Ivanov Viktor

#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

#include <cassert>
#include <cstddef>
#include <cstdint>
#include <cstring>
#include <type_traits>

#include "JoshUpscale/core.h"
#include "JoshUpscale/core/hip/hip_runtime.h"
#include "JoshUpscale/core/cuda_convert.h"
#include "JoshUpscale/core/tensor.h"
#include "JoshUpscale/core/utils.h"

namespace JoshUpscale {

namespace core {

namespace cuda {

namespace detail {

namespace {

template <class T2, class T1>
__device__ T2 cuda_bit_cast(T1 t1) {
	T2 t2{};
	std::memcpy(&t2, &t1, sizeof(T1));
	return t2;
}

template <typename T, typename StorageType>
struct BaseDataTypeTraits {
	using storage_type = StorageType;

	__device__ static storage_type store(T val1, T val2, T val3) {
		storage_type val{};
		val.x = val1;
		val.y = val2;
		val.z = val3;
		return val;
	}
	__device__ static T get(T val) {
		return val;
	}
};

template <typename T>
struct DataTypeTraits;

template <>
struct DataTypeTraits<std::uint8_t> : BaseDataTypeTraits<std::uint8_t, uchar4> {
};

template <>
struct DataTypeTraits<__half> {
	using storage_type = ushort3;

	__device__ static ushort3 store(__half val1, __half val2, __half val3) {
		return make_ushort3(cuda_bit_cast<std::uint16_t>(val1),
		    cuda_bit_cast<std::uint16_t>(val2),
		    cuda_bit_cast<std::uint16_t>(val3));
	}

	__device__ static __half get(std::uint16_t val) {
		return cuda_bit_cast<__half>(val);
	}
};

template <>
struct DataTypeTraits<float> : BaseDataTypeTraits<float, float3> {};

template <typename From, typename To>
struct CastTraits {
	__device__ static To convert(From value) {
		return static_cast<To>(value);
	}
};

template <typename T>
struct CastTraits<T, T> {};

template <>
struct CastTraits<std::uint8_t, __half> {
	__device__ static __half convert(std::uint8_t value) {
		return static_cast<__half>(static_cast<int>(value));
	}
};

constexpr unsigned int kBlockSize = 512;

template <typename From, typename To>
__global__ void castKernel(From *from, To *to, unsigned int numElements) {
	auto idx = (blockIdx.x * kBlockSize) + threadIdx.x;
	if (idx >= numElements) {
		return;
	}
	using src_storage = DataTypeTraits<From>::storage_type;
	using dst_storage = DataTypeTraits<To>::storage_type;
	src_storage src = reinterpret_cast<src_storage *>(from)[idx];
	reinterpret_cast<dst_storage *>(to)[idx] = DataTypeTraits<To>::store(
	    CastTraits<From, To>::convert(DataTypeTraits<From>::get(src.x)),
	    CastTraits<From, To>::convert(DataTypeTraits<From>::get(src.y)),
	    CastTraits<From, To>::convert(DataTypeTraits<From>::get(src.z)));
}

#define DECLARE_SPEC(From, To)                     \
	template __global__ void castKernel<From, To>( \
	    From * from, To * to, unsigned int numElements)  // NOLINT

DECLARE_SPEC(std::uint8_t, float);
DECLARE_SPEC(float, std::uint8_t);
DECLARE_SPEC(__half, std::uint8_t);
DECLARE_SPEC(std::uint8_t, __half);
DECLARE_SPEC(__half, float);
DECLARE_SPEC(float, __half);

#undef DECLARE_SPEC

template <typename T>
struct UnmanagedCudaBuffer {
	std::size_t size;
	T *ptr;

	std::size_t getByteSize() const {
		return size * sizeof(T);
	}

	std::size_t getNumElements() const {
		if constexpr (std::is_same_v<T, std::uint8_t>) {
			return size / 4;
		} else {
			return size / 3;
		}
	}

	UnmanagedCudaBuffer(std::size_t size, T *ptr) : size{size}, ptr{ptr} {
	}
	explicit UnmanagedCudaBuffer(const CudaBuffer<T> &s)
	    : UnmanagedCudaBuffer(s.getSize(), s.get()) {
	}
	explicit UnmanagedCudaBuffer(const CudaBuffer<DynamicType> &s)
	    : UnmanagedCudaBuffer(s.getSize(), reinterpret_cast<T *>(s.get())) {
	}
	template <typename Enable = std::enable_if<std::is_same_v<T, std::uint8_t>>>
	explicit UnmanagedCudaBuffer(const CudaTensor &s)
	    : UnmanagedCudaBuffer(s.getSize(), s.data()) {
		assert(s.isPlain());
	}
};

template <typename T>
struct ContainerTraits;

template <typename T>
struct ContainerTraits<CudaBuffer<T>> {
	using dataType = T;
};

template <>
struct ContainerTraits<CudaBuffer<DynamicType>> {};

template <>
struct ContainerTraits<CudaTensor> {
	using dataType = std::uint8_t;
};

template <typename T>
struct ContainerTraits<UnmanagedCudaBuffer<T>> {
	using dataType = T;
};

template <typename T, typename DataType = ContainerTraits<T>::dataType>
UnmanagedCudaBuffer<DataType> toUnmanaged(const T &s) {
	return UnmanagedCudaBuffer<DataType>(s);
}

template <typename From, typename To>
void cudaCastUnmanaged(const UnmanagedCudaBuffer<From> &from,
    const UnmanagedCudaBuffer<To> &to, const CudaStream &stream) {
	assert(from.getNumElements() == to.getNumElements());
	auto numElements = static_cast<unsigned int>(to.getNumElements());
	assert(numElements % WARP_SIZE == 0);
	auto numBlocks =
	    static_cast<unsigned int>((numElements + kBlockSize - 1) / kBlockSize);
	castKernel<From, To>
	    <<<numBlocks, kBlockSize, 0, stream>>>(from.ptr, to.ptr, numElements);
	cudaCheck(::hipGetLastError());
}

template <typename From, typename To>
void cudaCast(const From &from, const To &to, const CudaStream &stream) {
	cudaCastUnmanaged(toUnmanaged(from), toUnmanaged(to), stream);
}

template <typename T>
void cudaCast(
    const CudaBuffer<T> &from, CudaBuffer<T> &to, const CudaStream &stream) {
	cudaCopy(from, to, stream);
}

template <typename T>
void cudaCast(const CudaBuffer<DynamicType> &from, const T &to,
    const CudaStream &stream) {
	using dataType = ContainerTraits<T>::dataType;
	switch (from.getDataType()) {
	case DataType::UINT8:
		if constexpr (std::is_same_v<dataType, std::uint8_t>) {
			cudaCopy(from, to, stream);
		} else {
			cudaCastUnmanaged(UnmanagedCudaBuffer<std::uint8_t>(from),
			    toUnmanaged(to), stream);
		}
		break;
	case DataType::HALF:
		if constexpr (std::is_same_v<dataType, __half>) {
			cudaCopy(from, to, stream);
		} else {
			cudaCastUnmanaged(
			    UnmanagedCudaBuffer<__half>(from), toUnmanaged(to), stream);
		}
		break;
	case DataType::FLOAT:
		if constexpr (std::is_same_v<dataType, float>) {
			cudaCopy(from, to, stream);
		} else {
			cudaCastUnmanaged(
			    UnmanagedCudaBuffer<float>(from), toUnmanaged(to), stream);
		}
		break;
	default:
		unreachable();
	}
}

template <typename T>
void cudaCast(const T &from, const CudaBuffer<DynamicType> &to,
    const CudaStream &stream) {
	using dataType = ContainerTraits<T>::dataType;
	switch (to.getDataType()) {
	case DataType::UINT8:
		if constexpr (std::is_same_v<dataType, std::uint8_t>) {
			cudaCopy(from, to, stream);
		} else {
			cudaCastUnmanaged(toUnmanaged(from),
			    UnmanagedCudaBuffer<std::uint8_t>(to), stream);
		}
		break;
	case DataType::HALF:
		if constexpr (std::is_same_v<dataType, __half>) {
			cudaCopy(from, to, stream);
		} else {
			cudaCastUnmanaged(
			    toUnmanaged(from), UnmanagedCudaBuffer<__half>(to), stream);
		}
		break;
	case DataType::FLOAT:
		if constexpr (std::is_same_v<dataType, float>) {
			cudaCopy(from, to, stream);
		} else {
			cudaCastUnmanaged(
			    toUnmanaged(from), UnmanagedCudaBuffer<float>(to), stream);
		}
		break;
	default:
		unreachable();
	}
}

void cudaCast(const CudaBuffer<DynamicType> &from,
    const CudaBuffer<DynamicType> &to, const CudaStream &stream) {
	if (from.getDataType() == to.getDataType()) {
		cudaCopy(from, to, stream);
		return;
	}
	switch (to.getDataType()) {
	case DataType::UINT8:
		cudaCast(from, UnmanagedCudaBuffer<std::uint8_t>(to), stream);
		break;
	case DataType::HALF:
		cudaCast(from, UnmanagedCudaBuffer<__half>(to), stream);
		break;
	case DataType::FLOAT:
		cudaCast(from, UnmanagedCudaBuffer<float>(to), stream);
		break;
	default:
		unreachable();
	}
}

}  // namespace

}  // namespace detail

template <typename From, typename To>
void cudaCast(const From &from, const To &to, const CudaStream &stream) {
	detail::cudaCast(from, to, stream);
}

#define DECLARE_SPEC(From, To)        \
	template void cudaCast<From, To>( \
	    const From &from, const To &to, const CudaStream &stream);

DECLARE_SPEC(CudaBuffer<std::uint8_t>, CudaBuffer<float>);
DECLARE_SPEC(CudaBuffer<float>, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(CudaBuffer<__half>, CudaBuffer<float>);
DECLARE_SPEC(CudaBuffer<float>, CudaBuffer<__half>);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, CudaBuffer<__half>);
DECLARE_SPEC(CudaBuffer<__half>, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(CudaTensor, CudaBuffer<float>);
DECLARE_SPEC(CudaTensor, CudaBuffer<__half>);
DECLARE_SPEC(CudaBuffer<__half>, CudaTensor);
DECLARE_SPEC(CudaBuffer<float>, CudaTensor);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<__half>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<float>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaTensor);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaBuffer<__half>, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaBuffer<float>, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaTensor, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<DynamicType>);

#undef DECLARE_SPEC

namespace detail {

namespace {

template <typename From, typename To>
struct CopyKind {};

template <typename T>
struct CopyKind<CpuTensor, CudaBuffer<T>> {
	static constexpr ::hipMemcpyKind value =
	    ::hipMemcpyKind::hipMemcpyHostToDevice;
};

template <typename T>
struct CopyKind<CudaTensor, CudaBuffer<T>> {
	static constexpr ::hipMemcpyKind value =
	    ::hipMemcpyKind::hipMemcpyDeviceToDevice;
};

template <typename T>
struct CopyKind<CudaBuffer<T>, CpuTensor> {
	static constexpr ::hipMemcpyKind value =
	    ::hipMemcpyKind::hipMemcpyDeviceToHost;
};

template <typename T>
struct CopyKind<CudaBuffer<T>, CudaTensor> {
	static constexpr ::hipMemcpyKind value =
	    ::hipMemcpyKind::hipMemcpyDeviceToDevice;
};

template <typename From, typename T>
void cudaCopy(
    const From &from, const CudaBuffer<T> &to, const CudaStream &stream) {
	auto copyKind = CopyKind<From, CudaBuffer<T>>::value;
	std::size_t size = from.getByteSize();
	assert(from.getSize() == size && to.getByteSize() == size);
	if constexpr (std::is_same_v<T, DynamicType>) {
		assert(to.getDataType() == DataType::UINT8);
	}
	if (from.isPlain()) {
		cudaCheck(
		    ::hipMemcpyAsync(to.get(), from.data(), size, copyKind, stream));
	} else {
		std::size_t lineLength = from.getWidth() * 4 * sizeof(std::byte);
		cudaCheck(::hipMemcpy2DAsync(to.get(), lineLength, from.data(),
		    static_cast<std::size_t>(from.getStride()), lineLength,
		    from.getHeight(), copyKind, stream));
	}
}

template <typename T>
void cudaCopy(const GraphicsResource &from, const CudaBuffer<T> &to,
    const CudaStream &stream) {
	auto array = from.get();
	::hipExtent extent;
	cudaCheck(::hipArrayGetInfo(nullptr, &extent, nullptr, array));
	assert(to.getByteSize() == extent.width * extent.height * extent.depth);
	std::size_t lineLength = extent.width * 4 * sizeof(std::byte);
	if constexpr (std::is_same_v<T, DynamicType>) {
		assert(to.getDataType() == DataType::UINT8);
	}
	cudaCheck(::hipMemcpy2DFromArrayAsync(to.get(), lineLength, array, 0, 0,
	    lineLength, extent.height, ::hipMemcpyDeviceToDevice, stream));
}

template <typename To, typename T>
void cudaCopy(
    const CudaBuffer<T> &from, const To &to, const CudaStream &stream) {
	auto copyKind = CopyKind<CudaBuffer<T>, To>::value;
	std::size_t size = to.getByteSize();
	assert(from.getByteSize() == size && to.getByteSize() == size);
	if constexpr (std::is_same_v<T, DynamicType>) {
		assert(from.getDataType() == DataType::UINT8);
	}
	if (to.isPlain()) {
		cudaCheck(
		    ::hipMemcpyAsync(to.data(), from.get(), size, copyKind, stream));
	} else {
		std::size_t lineLength = to.getWidth() * 4 * sizeof(std::byte);
		cudaCheck(::hipMemcpy2DAsync(to.data(),
		    static_cast<std::size_t>(to.getStride()), from.get(), lineLength,
		    lineLength, to.getHeight(), copyKind, stream));
	}
}

template <typename T>
void cudaCopy(const CudaBuffer<T> &from, const GraphicsResource &to,
    const CudaStream &stream) {
	auto array = to.get();
	::hipExtent extent;
	cudaCheck(::hipArrayGetInfo(nullptr, &extent, nullptr, array));
	std::size_t lineLength = extent.width * 4 * sizeof(std::byte);
	assert(from.getByteSize() == extent.width * extent.height * extent.depth);
	if constexpr (std::is_same_v<T, DynamicType>) {
		assert(from.getDataType() == DataType::UINT8);
	}
	cudaCheck(::hipMemcpy2DToArrayAsync(array, 0, 0, from.get(), lineLength,
	    lineLength, extent.height, ::hipMemcpyDeviceToDevice, stream));
}

template <typename T>
[[noreturn]] void cudaCopy([[maybe_unused]] const CudaBuffer<DynamicType> &from,
    [[maybe_unused]] const UnmanagedCudaBuffer<T> &to,
    [[maybe_unused]] const CudaStream &stream) {
	unreachable();
}

template <typename From, typename To,
    typename Enable = std::enable_if<std::is_same_v<From, DynamicType> ||
                                     std::is_same_v<To, DynamicType> ||
                                     std::is_same_v<From, To>>>
void cudaCopy(const CudaBuffer<From> &from, const CudaBuffer<To> &to,
    const CudaStream &stream) {
	std::size_t size = from.getByteSize();
	assert(from.getByteSize() == size && to.getByteSize() == size);
	if constexpr (std::is_same_v<From, DynamicType> &&
	              std::is_same_v<To, DynamicType>) {
		assert(from.getDataType() == to.getDataType());
	}
	cudaCheck(::hipMemcpyAsync(to.get(), from.get(), size,
	    ::hipMemcpyKind::hipMemcpyHostToDevice, stream));
}

template <typename T>
void cudaCopy(const CudaBuffer<T> &from, const GenericTensor &to,
    const CudaStream &stream) {
	switch (to.getLocation()) {
	case DataLocation::CPU:
		cudaCopy(from, static_cast<const CpuTensor &>(to), stream);
		return;
	case DataLocation::CUDA:
		cudaCopy(from, static_cast<const CudaTensor &>(to), stream);
		return;
	default:
		unreachable();
	}
}

template <typename T>
void cudaCopy(const GenericTensor &from, const CudaBuffer<T> &to,
    const CudaStream &stream) {
	switch (from.getLocation()) {
	case DataLocation::CPU:
		cudaCopy(static_cast<const CpuTensor &>(from), to, stream);
		return;
	case DataLocation::CUDA:
		cudaCopy(static_cast<const CudaTensor &>(from), to, stream);
		return;
	default:
		unreachable();
	}
}

}  // namespace

}  // namespace detail

template <typename From, typename To>
void cudaCopy(const From &from, const To &to, const CudaStream &stream) {
	detail::cudaCopy(from, to, stream);
}

#define DECLARE_SPEC(From, To)        \
	template void cudaCopy<From, To>( \
	    const From &from, const To &to, const CudaStream &stream);

DECLARE_SPEC(CpuTensor, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(CudaTensor, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(GenericTensor, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(GraphicsResource, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(CpuTensor, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaTensor, CudaBuffer<DynamicType>);
DECLARE_SPEC(GenericTensor, CudaBuffer<DynamicType>);
DECLARE_SPEC(GraphicsResource, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, CpuTensor);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, CudaTensor);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, GenericTensor);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, GraphicsResource);
DECLARE_SPEC(CudaBuffer<DynamicType>, CpuTensor);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaTensor);
DECLARE_SPEC(CudaBuffer<DynamicType>, GenericTensor);
DECLARE_SPEC(CudaBuffer<DynamicType>, GraphicsResource);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(CudaBuffer<float>, CudaBuffer<float>);
DECLARE_SPEC(CudaBuffer<__half>, CudaBuffer<__half>);
DECLARE_SPEC(CudaBuffer<std::uint8_t>, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaBuffer<float>, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaBuffer<__half>, CudaBuffer<DynamicType>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<std::uint8_t>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<float>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<__half>);
DECLARE_SPEC(CudaBuffer<DynamicType>, CudaBuffer<DynamicType>);

#undef DECLARE_SPEC

}  // namespace cuda

}  // namespace core

}  // namespace JoshUpscale
